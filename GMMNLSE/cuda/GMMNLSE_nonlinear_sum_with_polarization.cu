
#include <hip/hip_runtime.h>
#define MAX_NUM_MODES 32 // the maximum number of modes for this cuda = sqrt(MaxThreadsPerBlock)
                         //                                           = sqrt(1024) for our Titan XP GPU

__global__ void GMMNLSE_nonlinear_sum_with_polarization(double2* Kerr, double2* Ra, double2* Rb,
                                                        const double2* A_t,
                                                        const double* SK,  const unsigned char* SK_nonzero_midx1234s,  const unsigned int* SK_beginning_nonzero,  const unsigned int* SK_ending_nonzero,
                                                        const double* SRa, const unsigned char* SRa_nonzero_midx1234s, const unsigned int* SRa_beginning_nonzero, const unsigned int* SRa_ending_nonzero,
                                                        const double* SRb, const unsigned char* SRb_nonzero_midx1234s, const unsigned int* SRb_beginning_nonzero, const unsigned int* SRb_ending_nonzero,
                                                        const bool include_Raman, const bool include_anisoRaman,
                                                        const unsigned int N, const unsigned int M,
                                                        const unsigned int NUM_MODES,
                                                        const unsigned int NUM_OPERATIONS) {
    const unsigned int midx1 = threadIdx.x / NUM_MODES;
    const unsigned int midx2 = threadIdx.x - midx1*NUM_MODES;

    const unsigned int NMIdx = blockIdx.x / NUM_OPERATIONS;
    const unsigned int OPERATIONIdx = blockIdx.x - NMIdx*NUM_OPERATIONS;

    const unsigned int Midx = NMIdx / N;
    const unsigned int Nidx = NMIdx - Midx*N;

    const unsigned int NM = N*M;
    const unsigned int NMMODES = NM*NUM_MODES;

    // Preload A_t to improve the performance (avoiding accessing the global memory too many times)
    __shared__ double2 this_A[MAX_NUM_MODES];
    if (midx1 == 0) this_A[midx2] = A_t[Nidx+Midx*N+midx2*NM];
    __syncthreads();

    const unsigned int this_SK_beginning_nonzero = SK_beginning_nonzero[midx2+midx1*NUM_MODES];
    const unsigned int this_SK_ending_nonzero = SK_ending_nonzero[midx2+midx1*NUM_MODES];
    const unsigned int this_SRa_beginning_nonzero = SRa_beginning_nonzero[midx2+midx1*NUM_MODES];
    const unsigned int this_SRa_ending_nonzero = SRa_ending_nonzero[midx2+midx1*NUM_MODES];
    const unsigned int this_SRb_beginning_nonzero = SRb_beginning_nonzero[midx2+midx1*NUM_MODES];
    const unsigned int this_SRb_ending_nonzero = SRb_ending_nonzero[midx2+midx1*NUM_MODES];

    unsigned int midx3, midx4;
    double a, b, c, d, e, f, pcdef, ncdef;
    switch (OPERATIONIdx) {
        case 0: // compute the Kerr term
            if (this_SK_beginning_nonzero > 0) {
                a = this_A[midx2].x;
                b = this_A[midx2].y;

                double2 this_Kerr;
                this_Kerr.x = 0; this_Kerr.y = 0; // initialized
                for (int i = this_SK_beginning_nonzero-1; i < this_SK_ending_nonzero-1; i++) {
                    midx3 = SK_nonzero_midx1234s[2+i*4]-1;
                    midx4 = SK_nonzero_midx1234s[3+i*4]-1;
            
                    c = this_A[midx3].x;
                    d = this_A[midx3].y;
                    e = this_A[midx4].x;
                    f = this_A[midx4].y;
            
                    pcdef = SK[i]*(c*e+d*f);
                    if (midx3 == midx4 || (int(midx3 & 1) != int(midx4 & 1)) ) {
                        if (midx3 == midx4) { // c=e, d=f --> ncdef=0
                            this_Kerr.x += a*pcdef;
                            this_Kerr.y += b*pcdef;
                        } else {
                            ncdef = SK[i]*(c*f-d*e);
                            this_Kerr.x += a*pcdef+b*ncdef;
                            this_Kerr.y += b*pcdef-a*ncdef;
                        }
                    } else {
                        this_Kerr.x += a*pcdef*2;
                        this_Kerr.y += b*pcdef*2;
                    }
                }
                Kerr[Nidx+Midx*N+midx1*NM+midx2*NMMODES] = this_Kerr;
            }
            break;

        case 1: // compute the SRa tensors, isotropic Raman response
            if (include_Raman && this_SRa_beginning_nonzero > 0) {
                double2 this_Ra;
                this_Ra.x = 0; this_Ra.y = 0; // initialized
                for (int i = this_SRa_beginning_nonzero-1; i < this_SRa_ending_nonzero-1; i++) {
                    midx3 = SRa_nonzero_midx1234s[2+i*4]-1;
                    midx4 = SRa_nonzero_midx1234s[3+i*4]-1;
        
                    c = this_A[midx3].x;
                    d = this_A[midx3].y;
                    e = this_A[midx4].x;
                    f = this_A[midx4].y;
            
                    if (midx3 == midx4 || (int(midx3 & 1) != int(midx4 & 1)) ) {
                        if (midx3 == midx4) { // c=e, d=f
                            this_Ra.x += SRa[i]*(c*e+d*f);
                        } else {
                            this_Ra.x += SRa[i]*(c*e+d*f);
                            this_Ra.y += SRa[i]*(d*e-c*f);
                        }
                    } else {
                        this_Ra.x += SRa[i]*(c*e+d*f)*2;
                    }
                }
                Ra[Nidx+Midx*N+midx1*NM+midx2*NMMODES] = this_Ra;
            }
            break;

        case 2: // compute the SRb tensors, anisotropic Raman response
            if (include_anisoRaman && this_SRb_beginning_nonzero > 0) {
                double2 this_Rb;
                this_Rb.x = 0; this_Rb.y = 0; // initialized
                for (int i = this_SRb_beginning_nonzero-1; i < this_SRb_ending_nonzero-1; i++) {
                    midx3 = SRb_nonzero_midx1234s[2+i*4]-1;
                    midx4 = SRb_nonzero_midx1234s[3+i*4]-1;
        
                    c = this_A[midx3].x;
                    d = this_A[midx3].y;
                    e = this_A[midx4].x;
                    f = this_A[midx4].y;
        
                    if (midx3 == midx4 || (int(midx3 & 1) != int(midx4 & 1)) ) {
                        if (midx3 == midx4) { // c=e, d=f
                            this_Rb.x += SRb[i]*(c*e+d*f);
                        } else {
                            this_Rb.x += SRb[i]*(c*e+d*f);
                            this_Rb.y += SRb[i]*(d*e-c*f);
                        }
                    } else {
                        this_Rb.x += SRb[i]*(c*e+d*f)*2;
                    }
                }
                Rb[Nidx+Midx*N+midx1*NM+midx2*NMMODES] = this_Rb;
            }
            break;
    }
}
